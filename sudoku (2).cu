#include "hip/hip_runtime.h"
/*
David & Yuki
11-08-2023

References:
Mentor Sessions:
  - told us to free cuda
  - helped us make sure we were changing our cells
  - helped give us the information needed about the predicate
213 Class:
  - negate method
  - bit functions
*/

#include <stdbool.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

#include "util.h"

// The width and height of a sudoku board
#define BOARD_DIM 9

// The width and height of a square group in a sudoku board
#define GROUP_DIM 3

// The number of boards to pass to the solver at one time
#define BATCH_SIZE 128


/**
 * A board is an array of 81 cells. Each cell is encoded as a 16-bit integer.
 * Read about this encoding in the documentation for the digit_to_cell and
 * cell_to_digit functions' documentation.
 *
 * Boards are stored as a one-dimensional array. It doesn't matter if you use
 * row-major or column-major form (that just corresponds to a rotation of the
 * sudoku board) but you will need to convert column and row to a single index
 * when accessing the board to propagate constraints.
 */
typedef struct board {
  uint16_t cells[BOARD_DIM * BOARD_DIM];
} board_t;

// Declare a few functions. Documentation is with the function definition.
void print_board(board_t* board);
__host__ __device__ uint16_t digit_to_cell(int digit);
__host__ __device__ int cell_to_digit(uint16_t cell);
__host__ __device__ int single_dig(board_t* curr_board, int i, int index, int predicate);
__host__ __device__ int get_index(int x, int y);


/**
 * This kernel runs on the GPU and uses loops to iterate over cells of a Sudoku board.
 *
 * \param boards      An array of boards that should be solved.
 */
__global__ void kernel(board_t* boards) {

  // Get the thread's column and row indices in the block.
  int col = threadIdx.y;
  int row = threadIdx.x;

  // Calculate region coordinates based on the GROUP_DIM size.
  int x = col / GROUP_DIM;
  int y = row / GROUP_DIM;

  int index = get_index(col, row);
  //set to 1 as an initialization
  int predicate = 1;
  board_t* current_board = &boards[blockIdx.x];

  // loops as long as long as there is a change
  while (__syncthreads_count(predicate) != 0) {
    //resets predicate to no changes
    predicate = 0;

    // If the current cell is already filled in, skip it.
    if (cell_to_digit(current_board->cells[index]) != 0) {
      break;
    }

    //loops cols
    for (int i = col; i <= (col + 72); i += BOARD_DIM) {
      predicate = single_dig(current_board, i, index, predicate);
    }

    //loops rows
    for (int j = (row * BOARD_DIM); j < (row + 1) * BOARD_DIM; j++) {
      predicate = single_dig(current_board, j, index, predicate);
    }

    // Check and update constraints in region
    for (int region_col = x * GROUP_DIM; region_col < (x + 1) * GROUP_DIM; region_col++) {
      for (int region_row = y * GROUP_DIM; region_row < (y + 1) * GROUP_DIM; region_row++) {
        int region_cell_index = get_index(region_col, region_row);
        predicate = single_dig(current_board, region_cell_index, index, predicate);
      }
    }
  }
}

/**
 * This is an error handling function. It handles CUDA errors by printing an error message and
 * terminating the program.
 *
 * \param error    The hipError_t error code returned from CUDA runtime calls.
 * \param errorcode   An integer code to print with the error message for easy identification.
 */
// Error handling function
void helper(hipError_t error, int errorCode) {
  //check if there are any errors and handles the error
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error %d: %s\n", errorCode, hipGetErrorString(error));
        exit(errorCode);
    }
}

/**
 * Take an array of boards and solve them all. The number of boards will be no
 * more than BATCH_SIZE, but may be less if the total number of input boards
 * is not evenly-divisible by BATCH_SIZE.
 *
 * TODO: Implement this function! You will need to add a GPU kernel, and you
 *       will almost certainly want to write helper functions; that is fine.
 *       However, you should not modify any other functions in this file.
 *
 * \param boards      An array of boards that should be solved.
 * \param num_boards  The numebr of boards in the boards array
 */
void solve_boards(board_t* boards, size_t num_boards) {

    board_t* gpuCpy = nullptr;
    // Allocate GPU memory for the boards.
    helper(hipMalloc(&gpuCpy, sizeof(board_t) * num_boards), 1);
    // Copy the boards from host to device memory.
    helper(hipMemcpy(gpuCpy, boards, sizeof(board_t) * num_boards, hipMemcpyHostToDevice), 2);

    int num_blocks = num_boards;
    dim3 threads_per_block(BOARD_DIM, BOARD_DIM);

    // Launch the kernel to solve the boards
    kernel<<<num_blocks, threads_per_block>>>(gpuCpy);
     // Check for any errors during kernel execution.
    helper(hipGetLastError(), 3);

    // Synchronize the device to ensure all boards are solved.
    helper(hipDeviceSynchronize(), 4);

    // Copy the solved boards back to host memory.
    helper(hipMemcpy(boards, gpuCpy, sizeof(board_t) * num_boards, hipMemcpyDeviceToHost), 2);
    // Free the GPU memory allocated for boards
    hipFree(gpuCpy);
}

/**
 * Checks if the cell we are looking at is a single digit and updates the possible values for a
 * target cell based on Sudoku constraints.
 *
 * \param current_board   A pointer to the current Sudoku board being solved.
 * \param current_cell_index The index of the current cell being checked.
 * \param target The index of the target cell that is being updated based on the current cell's
 * value.
 * \param predicate An integer flag that tracks whether changes have been made. It increments when
 * a cell's possible values are updated.
 */
__host__ __device__ int single_dig(board_t* current_board, int current_cell_index, int target, int predicate) {
  // Ensure that the current cell is not checking itself
  if (current_cell_index == target) {
    return predicate;  // No changes needed, return the current predicate
  }

  // Extract the digit from the current cell (if reduced to one digit)
  int digit = cell_to_digit(current_board->cells[current_cell_index]);

  if (digit > 0) {
    //Idea taken from class
    uint16_t negate = ~(1 << digit);

    // Update the target cell's constraints by applying the mask
    uint16_t target_cell_constraints = current_board->cells[target] & negate;

    // Check if constraints were updated
    if (target_cell_constraints != current_board->cells[target_cell_index]) {
      // Update the target cell with the new constraints
      current_board->cells[target_cell_index] = target_cell_constraints;

      // Increment the predicate to track changes
      predicate++;
    }
  }

  return predicate;  // Return the updated predicate
}


/**
 * A simple helper function to calculate the 1D index from 2D coordinates, which is often necessary
 * when working with 2D data in a 1D array in CUDA.
 *
 * \param x the row index
 * \param y the column index
 */

__host__ __device__ int get_index(int x, int y) {
  return (x + (BOARD_DIM * y));
}  // get_index

/**
 * Take as input an integer value 0-9 (inclusive) and convert it to the encoded
 * cell form used for solving the sudoku. This encoding uses bits 1-9 to
 * indicate which values may appear in this cell.
 *
 * For example, if bit 3 is set to 1, then the cell may hold a three. Cells that
 * have multiple possible values will have multiple bits set.
 *
 * The input digit 0 is treated specially. This value indicates a blank cell,
 * where any value from one to nine is possible.
 *
 * \param digit   An integer value 0-9 inclusive
 * \returns       The encoded form of digit using bits to indicate which values
 *                may appear in this cell.
 */
__host__ __device__ uint16_t digit_to_cell(int digit) {
  if (digit == 0) {
    // A zero indicates a blank cell. Numbers 1-9 are possible, so set bits 1-9.
    return 0x3FE;
  } else {
    // Otherwise we have a fixed value. Set the corresponding bit in the board.
    return 1 << digit;
  }
}

/*
 * Convert an encoded cell back to its digit form. A cell with two or more
 * possible values will be encoded as a zero. Cells with one possible value
 * will be converted to that value.
 *
 * For example, if the provided cell has only bit three set, this function will
 * return the value 3.
 *
 * \param cell  An encoded cell that uses bits to indicate which values could
 *              appear at this point in the board.
 * \returns     The value that must appear in the cell if there is only one
 *              possibility, or zero otherwise.
 */
__host__ __device__ int cell_to_digit(uint16_t cell) {
  // Get the index of the least-significant bit in this cell's value
#if defined(__CUDA_ARCH__)
  int msb = __clz(cell);
  int lsb = sizeof(unsigned int) * 8 - msb - 1;
#else
  int lsb = __builtin_ctz(cell);
#endif

  // Is there only one possible value for this cell? If so, return it.
  // Otherwise return zero.
  if (cell == 1 << lsb)
    return lsb;
  else
    return 0;
}

/**
 * Read in a sudoku board from a string. Boards are represented as an array of
 * 81 16-bit integers. Each integer corresponds to a cell in the board. Bits
 * 1-9 of the integer indicate whether the values 1, 2, ..., 8, or 9 could
 * appear in the given cell. A zero in the input indicates a blank cell, where
 * any value could appear.
 *
 * \param output  The location where the board will be written
 * \param str     The input string that encodes the board
 * \returns       true if parsing succeeds, false otherwise
 */
bool read_board(board_t* output, const char* str) {
  for (int index = 0; index < BOARD_DIM * BOARD_DIM; index++) {
    if (str[index] < '0' || str[index] > '9') return false;

    // Convert the character value to an equivalent integer
    int value = str[index] - '0';

    // Set the value in the board
    output->cells[index] = digit_to_cell(value);
  }

  return true;
}

/**
 * Print a sudoku board. Any cell with a single possible value is printed. All
 * cells with two or more possible values are printed as blanks.
 *
 * \param board   The sudoku board to print
 */
void print_board(board_t* board) {
  for (int row = 0; row < BOARD_DIM; row++) {
    // Print horizontal dividers
    if (row != 0 && row % GROUP_DIM == 0) {
      for (int col = 0; col < BOARD_DIM * 2 + BOARD_DIM / GROUP_DIM; col++) {
        printf("-");
      }
      printf("\n");
    }

    for (int col = 0; col < BOARD_DIM; col++) {
      // Print vertical dividers
      if (col != 0 && col % GROUP_DIM == 0) printf("| ");

      // Compute the index of this cell in the board array
      int index = col + row * BOARD_DIM;

      // Get the index of the least-significant bit in this cell's value
      int digit = cell_to_digit(board->cells[index]);

      // Print the digit if it's not a zero. Otherwise print a blank.
      if (digit != 0)
        printf("%d ", digit);
      else
        printf("  ");
    }
    printf("\n");
  }
  printf("\n");
}

/**
 * Check through a batch of boards to see how many were solved correctly.
 *
 * \param boards        An array of (hopefully) solved boards
 * \param solutions     An array of solution boards
 * \param num_boards    The number of boards and solutions
 * \param solved_count  Output: A pointer to the count of solved boards.
 * \param error:count   Output: A pointer to the count of incorrect boards.
 */
void check_solutions(board_t* boards,
                     board_t* solutions,
                     size_t num_boards,
                     size_t* solved_count,
                     size_t* error_count) {
  // Loop over all the boards in this batch
  for (int i = 0; i < num_boards; i++) {
    // Does the board match the solution?
    if (memcmp(&boards[i], &solutions[i], sizeof(board_t)) == 0) {
      // Yes. Record a solved board
      (*solved_count)++;
    } else {
      // No. Make sure the board doesn't have any constraints that rule out
      // values that are supposed to appear in the solution.
      bool valid = true;
      for (int j = 0; j < BOARD_DIM * BOARD_DIM; j++) {
        if ((boards[i].cells[j] & solutions[i].cells[j]) == 0) {
          valid = false;
        }
      }

      // If the board contains an incorrect constraint, record an error
      if (!valid) (*error_count)++;
    }
  }
}

/**
 * Entry point for the program
 */
int main(int argc, char** argv) {
  // Check arguments
  if (argc != 2) {
    fprintf(stderr, "Usage: %s <input file name>\n", argv[0]);
    exit(1);
  }

  // Try to open the input file
  FILE* input = fopen(argv[1], "r");
  if (input == NULL) {
    fprintf(stderr, "Failed to open input file %s.\n", argv[1]);
    perror(NULL);
    exit(2);
  }

  // Keep track of total boards, boards solved, and incorrect outputs
  size_t board_count = 0;
  size_t solved_count = 0;
  size_t error_count = 0;

  // Keep track of time spent solving
  size_t solving_time = 0;

  // Reserve space for a batch of boards and solutions
  board_t boards[BATCH_SIZE];
  board_t solutions[BATCH_SIZE];

  // Keep track of how many boards we've read in this batch
  size_t batch_count = 0;

  // Read the input file line-by-line
  char* line = NULL;
  size_t line_capacity = 0;
  while (getline(&line, &line_capacity, input) > 0) {
    // Read in the starting board
    if (!read_board(&boards[batch_count], line)) {
      fprintf(stderr, "Skipping invalid board...\n");
      continue;
    }

    // Read in the solution board
    if (!read_board(&solutions[batch_count], line + BOARD_DIM * BOARD_DIM + 1)) {
      fprintf(stderr, "Skipping invalid board...\n");
      continue;
    }

    // Move to the next index in the batch
    batch_count++;

    // Also increment the total count of boards
    board_count++;

    // If we finished a batch, run the solver
    if (batch_count == BATCH_SIZE) {
      size_t start_time = time_ms();
      solve_boards(boards, batch_count);
      solving_time += time_ms() - start_time;

      check_solutions(boards, solutions, batch_count, &solved_count, &error_count);

      // Reset the batch count
      batch_count = 0;
    }
  }

  // Check if there's an incomplete batch to solve
  if (batch_count > 0) {
    size_t start_time = time_ms();
    solve_boards(boards, batch_count);
    solving_time += time_ms() - start_time;

    check_solutions(boards, solutions, batch_count, &solved_count, &error_count);
  }

  // Print stats
  double seconds = (double)solving_time / 1000;
  double solving_rate = (double)solved_count / seconds;

  // Don't print nan when solver is not implemented
  if (seconds < 0.01) solving_rate = 0;

  printf("Boards: %lu\n", board_count);
  printf("Boards Solved: %lu\n", solved_count);
  printf("Errors: %lu\n", error_count);
  printf("Total Solving Time: %lums\n", solving_time);
  printf("Solving Rate: %.2f sudoku/second\n", solving_rate);

  return 0;
}
